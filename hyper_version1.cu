#include "hip/hip_runtime.h"
#include <mat.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <iostream>
#include "hipblas.h"
#include "cokus.cpp"
#include "cuda_util.h"
#include <hip/hip_runtime.h>
using namespace std;

const int KER_NUM = 20;//卷积核数量
const int P_NUM = 3;//每次卷积的层数
const int LEAP = 2;//跳数
const int GP_NUM = 5;//maxpooling每组的个数
const int NEU_NUM1 = 100;
const int NEU_NUM2 = 16;//输出层神经元个数
const int NEIGHBOR = 8;//定义邻居个数
const double LEARN_RATE = 0.01;
//const int DATA_BATCH = 512;//每次处理512个像素对应的数据

//CUDA初始化
bool InitCUDA(){
	int count;
	hipGetDeviceCount(&count);
	if(count==0){
		fprintf(stderr,"There is no device.\n");
		return false;
	}
	int i;
	for (i =0; i<count;i++){
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop,i)==hipSuccess){
			if(prop.major>=1){                                                                                                                                      break;
			}
		}
	}
	if(i==count){
		fprintf(stderr,"There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}


//copy数据到shared memory
__device__ void copy_data_to_shared(double * data, double * data_tmp,int head, int length){
	for(int i=0; i<length; i++){
		data_tmp[i] = data[i+head];
	}

	__syncthreads();
}

//GPU端负责卷积
__global__ static void convol(int iter,int i0,double * train,double * kernel,double * re,int x,int y,int z,int re_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;//保存当前线程编号

	//每个线程负责一个卷积核与一个3*3*hight柱状图像的卷积
	if (id < KER_NUM){
		extern __shared__ double train_tmp[];
		//__shared__ double train_tmp[9*200];
		int st = i0 * x * y * z;

		copy_data_to_shared(train,train_tmp,st,x*y*z);//复制train到shared memory中

		/*double * ker = new double [x*y*P_NUM];//载入对应的kernel到寄存器
		for(int i=0; i<x*y*P_NUM; i++){
			ker[i] = kernel[id*x*y*P_NUM + i];
		}*/

		int dim_x = 0, dim_y = 0, dim_z = 0;//初始位置为(0,0,0)

		double mid;
		int i_1=0;
		for(; dim_z+P_NUM-1 < z; dim_z=dim_z+LEAP){//每次跳LEAP层
			mid = 0.0;

			for(int i_0=0;i_0<P_NUM;i_0++){//每次进行3*3*P_NUM的像素块的卷积
				for(int i=0;i<x;i++){
					for(int j=0;j<y;j++){
						mid =mid + train_tmp[dim_x+j + (dim_y+i) * x + (dim_z+i_0)*x*y] * kernel[j + i*x + i_0*x*y + id*x*y*P_NUM];
					}
				}
			}

			re[i_1 + id * re_size] =2 * (1/(1 + 1/exp(2*mid))) - 1;//激活函数tanh
			i_1 ++;
		}

	}
}

//GPU端进行下采样
__global__ static void maxpooling(int iter,double * re,double * mre,int * mre_index,int re_size,int mre_num){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
       	int id = tid + iter * threadNum; 
	
	//int res = re_size, mres = mre_num;
	//extern __shared__ double re_tmp[];
	//copy_data_to_shared(re, re_tmp, 0, re_size*KER_NUM);

	if(id < KER_NUM){
		double mid;
		int mid_index;
		for(int i=0; i<mre_num; i++){
			mid = re[i*GP_NUM + id*re_size];//存放每组第一个值
			mid_index = i*GP_NUM + id*re_size;
			for(int j=i*GP_NUM+1; j<(i+1)*GP_NUM && j<re_size; j++){
				if(mid < re[j + id*re_size]){
					mid = re[j + id*re_size];
					mid_index = j+id*re_size;
				}
			}
			mre[i + id * mre_num] = mid;
			mre_index[i + id * mre_num] = mid_index;
		}
	}
}

//全连接层,每个线程负责一个神经元输出结果的计算
__global__ static void fullconnect(int iter,double * mre,double * omega,double * bias,double * F1,int mre_size){
	int tid = blockIdx.x * blockDim.x +threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;

	if(id < NEU_NUM1){
		//复制mre数组到共享内存
		//__shared__ double mre_tmp[50 * KER_NUM];
	        extern __shared__ double mre_tmp[];	
		copy_data_to_shared(mre,mre_tmp,0,mre_size);
		
		//计算神经元的输出
		double mid=0;
		int j; 
		for(int i=id*mre_size; i<(id+1)*mre_size; i++){
			j = 0;
			mid = mid + omega[i] * mre_tmp[j];
			j = j + 1;
		}
		F1[id] = 1/(1 + 1/exp(mid + bias[id]));//激活函数sigmod
	}
}

//输出层，每个线程负责一个神经元输出结果的计算
__global__ static void output(int iter, double * F1, double * omega2, double * bias, double * O2){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;

	if(id < NEU_NUM2){
		//复制F1到共享内存中
		__shared__ double F1_tmp[NEU_NUM1];
		copy_data_to_shared(F1, F1_tmp, 0, NEU_NUM1);

		//计算神经元的输出
		double mid = 0;
		int j;
		for(int i=id*NEU_NUM1; i<(id+1)*NEU_NUM1; i++){
			j = 0;
			mid = mid + omega2[i] * F1_tmp[j];
			j = j + 1;
		}
		O2[id] = 1/(1 + 1/exp(mid + bias[id]));//激活函数sigmod
	}
}

/*反向传播*/
//输出层
__global__ static void bp_output(int iter,int train_idx, double * labels, double * O2, double * bias2, double * delta_L_a, double * delta_L_z)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;
	if(id < NEU_NUM2){
		delta_L_a[id] = -(labels[id + train_idx * NEU_NUM2] - O2[id]);
		delta_L_z[id] = delta_L_a[id] * O2[id] *(1 - O2[id]);
		bias2[id] = bias2[id] - delta_L_z[id]*LEARN_RATE;
	}
}
//全连接层
__global__ static void bp_fullconnect(int iter, double * omega2,double * bias1, double * F1, double * delta_L_a, double * delta_L_z, double *delta_f_a, double * delta_f_z)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;
	if(id < NEU_NUM1){
		double mid = 0;
		double delta_f_w;

		for(int i=0; i<NEU_NUM2; i++){
			mid = mid + omega2[i + id*NEU_NUM2] * delta_L_z[i];
			//delta_f_b[i] = delta_L_z[i];
			delta_f_w = F1[id] * delta_L_z[i];

			omega2[i + id*NEU_NUM2] = omega2[i + id*NEU_NUM2] - LEARN_RATE * delta_f_w;
			//bias2[i] = bias2[i] - LEARN_RATE*delta_f_b[i];
		}
		delta_f_a[id] = mid;
		delta_f_z[id] = delta_f_a[id] * F1[id] * (1 - F1[id]);
		bias1[id] = bias1[id] - LEARN_RATE * delta_f_z[id];
	}
}
//maxpooling层（卷积层）
__global__ static void bp_maxpooling(int iter, int mre_size,int *mre_index, double * omega1,double *mre, double * delta_f_a, double * delta_f_z, double * delta_m_a, double * delta_2, double * delta_22)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
        int threadNum = blockDim.x * gridDim.x;
        int id = tid + iter * threadNum;
	if(id < mre_size){
		double mid = 0;
		double delta_m_w;
		for(int i=0; i<NEU_NUM1; i++){
			mid = mid + omega1[i + id*NEU_NUM1] * delta_f_z[i];
			//delta_2[i + id*NEU_NUM1] = mid;
			delta_m_w = mre[id] * delta_f_z[i];
			omega1[i + id*NEU_NUM1] = omega1[i + id*NEU_NUM1] - LEARN_RATE * delta_m_w;
		}
		delta_m_a[id] = mid;
		delta_2[id] = delta_m_a[id];
		
		//int idx = mre_index[id];
		delta_22[mre_index[id]] = delta_2[id];
	}
}
//输入层
/*__global__ static void bp_map_convol(int iter,int mre_num, int * max_idx,double * re, double * delta_2, double * delta_22){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;
	if(id < mre_num * KER_NUM){
		int idx = max_idx[id];
		for(int i=0; i<NEU_NUM1; i++){
			delta_22[i + idx*NEU_NUM1] = delta_2[i +id*NEU_NUM1];
			delta_22[i + idx*NEU_NUM1] = delta_22[i + idx*NEU_NUM1] * (1 - re[idx]) * (1 + re[idx]);
		}

		extern __shared__ double train_tmp[];
		copy_data_to_shared(train_tmp, data, 9*z*train_idx, 9*z);

		double delta_z = 0;
		double delta_w = 0;
		int ii=0;
		for(int i=0; i<re_size; i++){//处理一个卷积核的所有计算结果
			ii = i*LEAP*9;
			for(int j=0; j<3*3*P_NUM; j++){
				delta_z = delta_m_a[i/GP_NUM + id*mre_num] * (1 - train_tmp[ii+j] * train_tmp[ii+j]);
				delta_w = delta_z * train_tmp[ii+j];
				kernel[j + id*9*P_NUM] = kernel[j + id*9*P_NUM] - LEARN_RATE*delta_w;
			}
		}
	}
}*/
//计算并更新kernel
__global__ static void bp_update_kernel(int iter,int i0, int x, int y, int z, int mre_num,int re_size, int * mre_index, double * delta_22, double * data, double * delta_k_w,double * kernel)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum =blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;
	if(id < KER_NUM){
		extern __shared__ double train_tmp[];
		copy_data_to_shared(data, train_tmp, x*y*z*i0, x*y*z);

		for (int i=0; i<mre_num; i++){
			int idx = mre_index[i + id*mre_num];
			int n = idx % re_size;//对应卷积的第n块数据
			int head = x*y*LEAP*n;
			for(int j=0; j<x*y*P_NUM; j++){
				delta_k_w[j] = delta_k_w[j] + /*delta_22[idx] **/ train_tmp[j+head];
			}
		
		}

		for(int i=0;i<x*y*P_NUM;i++){
			delta_k_w[i] = delta_k_w[i]/mre_num;
			kernel[id*x*y*P_NUM+i] = kernel[id*x*y*P_NUM+i] - LEARN_RATE*delta_k_w[i];
		}
		//double delta_k_b = delta_22[id + idx*re_size*NEU_NUM1];
	}
}

//数据预处理
__global__ static void processing(int iter, double * data, int * train_index, double * processed_data, int x, int y, int z, int train_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;

	//int idx = id * (NEIGHBOR+1) * z;//记录processed_data的开始位置
	if (id < train_size){
		int idx = id * (NEIGHBOR+1) * z;
		for (int i=0; i<z; i++){
			for (int j=0; j<(NEIGHBOR+1); j++){
				processed_data[idx] = data[train_index[j + id*(NEIGHBOR+1)] + i * x*y];
				idx = idx + 1;	
			}
		}
	}
}

double lossfunction(double * output, double * labels, int idx){
	double l = 0;
	for(int i=0; i<NEU_NUM2; i++){
		l = l + (output[i] - labels[i + idx*NEU_NUM2]) * (output[i] - labels[i + idx*NEU_NUM2]);
	}
	l = l/2;
	return l;
}

//训练
int training(double * data, double * labels, int x, int y, int z){
	double * gpu_data;//显存上存储原始数据
	double * gpu_processed_train;//显存上存储处理之后的数据
	int * gpu_train_index;//训练数据的索引
	double * gpu_processed_labels;

	//计算有标签像素的个数
	int train_size = 0;
	int * train_index = new int [x*y * (NEIGHBOR + 1)];//9行，x*y列。每列保存一个像素及其邻居的索引位置
	double * processed_labels = new double [x*y * NEU_NUM2];
	for (int i=0; i<x*y; i++){
		if (labels[i] != 0){
			train_index[(NEIGHBOR/2) + train_size * (NEIGHBOR+1)] = i;//当前像素索引
			train_index[(NEIGHBOR/2) + train_size * (NEIGHBOR+1) - 1] = i - 1;
			train_index[(NEIGHBOR/2) + train_size * (NEIGHBOR+1) + 1] = i + 1;
			for(int j0=0;j0<3;j0++){
				train_index[j0 + train_size * (NEIGHBOR+1)] = i - 1 - x + j0;
				train_index[j0+6 + train_size * (NEIGHBOR+1)] = i - 1 + x + j0;
			}

			if((i % x) == 0){//第一行
				for (int j=0; j<3; j++)
					train_index[j*3 + train_size*(NEIGHBOR+1)] = train_index[j*3+2 + train_size*(NEIGHBOR+1)];
			}
			if((i % x) == (x-1)){//最后一行
				for(int j=0;j<3;j++)
		       			train_index[j*3+2 + train_size*(NEIGHBOR+1)] = train_index[j*3 + train_size*(NEIGHBOR+1)];
			}
			if((i/x) == 0){//第一列
				for(int j=0;j<3;j++)
					train_index[j + train_size*(NEIGHBOR+1)] = train_index[j+6 + train_size*(NEIGHBOR+1)];
			}
			if((i/x) == (y-1)){//最后一列
				for(int j=0;j<3;j++)
					train_index[j+6  + train_size*(NEIGHBOR+1)] = train_index[j + train_size*(NEIGHBOR+1)];
			}

			int mid = int(labels[i])-1 + train_size*NEU_NUM2;
			processed_labels[mid] = 1;
			train_size = train_size + 1;
		}
	}
	fprintf(stdout,"train_size:%d\n",train_size);
	fprintf(stdout,"train_index[0]:%d %d %d %d,%d %d %d %d\n",train_index[0],train_index[1],train_index[2],train_index[3],train_index[5],train_index[6],train_index[7],train_index[8]);
	//fprintf(stdout,"train_index[5211]:%d %d %d %d,%d %d %d %d\n",train_index[46890],train_index[46891],train_index[46892],train_index[46893],train_index[46895],train_index[46896],train_index[46897],train_index[46898]);
	
	//int * train_index = new int [train_size * (NEIGHBOR + 1)];//train_size列，9行。每行保存一个像素及其邻居的索引位置


	//分配显存，拷贝数据到显存上
	SAFE_CALL(hipMalloc((void **) &gpu_data, sizeof(double) * x * y * z));
	SAFE_CALL(hipMemcpy(gpu_data, data, sizeof(double)* x * y * z, hipMemcpyHostToDevice));

	SAFE_CALL(hipMalloc((void **) &gpu_train_index, sizeof(int) * train_size * (NEIGHBOR+1)));
	SAFE_CALL(hipMemcpy(gpu_train_index, train_index, sizeof(int) * train_size * (NEIGHBOR+1), hipMemcpyHostToDevice));

	SAFE_CALL(hipMalloc((void **) &gpu_processed_train, sizeof(double) * train_size * (NEIGHBOR+1) * z));//每一批数据的大小

	int gridsize = 64;
	int blocksize = 1024;
	int threadNum = gridsize * blocksize; 
	//double * processed_train = new double [train_size * (NEIGHBOR+1) * z];
	//预处理
	for (int iter=0; iter<=train_size/threadNum; iter++){
		processing<<<gridsize,blocksize>>>(iter, gpu_data, gpu_train_index, gpu_processed_train, x, y, z, train_size);
	}
	hipDeviceSynchronize();
	//SAFE_CALL(hipMemcpy(processed_train, gpu_processed_train, sizeof(double) * train_size * (NEIGHBOR+1) * z, hipMemcpyDeviceToHost));
	SAFE_CALL(hipFree(gpu_data));
	SAFE_CALL(hipFree(gpu_train_index));
	hipDeviceSynchronize();
	//fprintf(stdout,"Processed train data:%lf %lf %lf %lf\n",processed_train[0],processed_train[1],processed_train[2],processed_train[3]);
	
	//前向传播
	double * kernel = new double [(NEIGHBOR+1)*P_NUM*KER_NUM];

	//随机生成kernekl数组
	for(int i=0; i<(NEIGHBOR+1)*P_NUM*KER_NUM; i++){
		kernel[i] = 2 * (rand()/(double)(RAND_MAX)) - 1 ;
		if(kernel[i] == 0 || kernel[i] == -1 || kernel[i] == 1)
			kernel[i] = 0.001;
	}
	
	//计算每次卷积的结果个数
	int re_size = 0;
	for (int i=0; i+P_NUM-1<z; i+=LEAP){
		re_size ++;
	}

	double * re = new double [re_size * KER_NUM];
	fprintf(stdout,"Size of re:%d\n",re_size);

	int mre_num = re_size/GP_NUM + 1;
	if(re_size/GP_NUM == 0){
		mre_num = re_size / GP_NUM;
	}
	fprintf(stdout,"mre_num:%d\n",mre_num);
	int mre_size = mre_num * KER_NUM;
	int ome_num1 = mre_num * KER_NUM * NEU_NUM1;//第一层网络的输入权重个数
	int ome_num2 = NEU_NUM1 * NEU_NUM2;//输出层的权重个数
	
	//double * gpu_labels;
	double * gpu_kernel;
	double * gpu_re;//存放卷积结果
	double * gpu_mre;//存放maxpooling结果
	int * gpu_mre_index;//存放每组最大值的索引
	double * gpu_omega1;//第一层网络的输入权重
	double * gpu_F1;//第一层神经元的输出
	double * gpu_bias1;
	double * gpu_omega2;
	double * gpu_O2;
	double * gpu_bias2;
	double * gpu_delta_La;
	double * gpu_delta_Lz;
	double * gpu_delta_fa;
	double * gpu_delta_fz;
	double * gpu_delta_ma;
	double * gpu_delta_mz;
	double * gpu_delta_2;
	double * gpu_delta_22;
	double * gpu_delta_kw;
	//double * gpu_delta_ia;
	//double * gpu_delta_iz;
	double * delta_22 = new double [re_size*KER_NUM];

	//复制标签
	SAFE_CALL(hipMalloc((void**) &gpu_processed_labels, sizeof(double) * train_size * NEU_NUM2));
	SAFE_CALL(hipMemcpy(gpu_processed_labels,processed_labels,sizeof(double) * train_size * NEU_NUM2,hipMemcpyHostToDevice));
	//复制随机初始化的kernel数组
	SAFE_CALL(hipMalloc((void**) &gpu_kernel,sizeof(double) * (NEIGHBOR+1) * P_NUM * KER_NUM));
	SAFE_CALL(hipMemcpy(gpu_kernel,kernel,sizeof(double) * (NEIGHBOR+1) * P_NUM * KER_NUM,hipMemcpyHostToDevice));
	//卷积结果存入gpu_re，分配显存
	SAFE_CALL(hipMalloc((void **) &gpu_re,sizeof(double) * re_size * KER_NUM));
	//输出层偏导数
	SAFE_CALL(hipMalloc((void **) &gpu_delta_La, sizeof(double) * NEU_NUM2));
	SAFE_CALL(hipMalloc((void **) &gpu_delta_Lz, sizeof(double) * NEU_NUM2));
	//全连接层偏导数
	SAFE_CALL(hipMalloc((void **) &gpu_delta_fa, sizeof(double) * NEU_NUM1));
	SAFE_CALL(hipMalloc((void **) &gpu_delta_fz, sizeof(double) * NEU_NUM1));
	//maxpooling
	SAFE_CALL(hipMalloc((void **) &gpu_delta_ma, sizeof(double) * mre_size));
	SAFE_CALL(hipMalloc((void **) &gpu_delta_mz, sizeof(double) * mre_size));
	//输入层
	SAFE_CALL(hipMalloc((void **) &gpu_delta_2, sizeof(double) * mre_size));
	SAFE_CALL(hipMalloc((void **) &gpu_delta_22,sizeof(double) * re_size * KER_NUM));
	SAFE_CALL(hipMemcpy(gpu_delta_22, delta_22, sizeof(double) * re_size * KER_NUM, hipMemcpyHostToDevice));
	SAFE_CALL(hipMalloc((void **) &gpu_delta_kw, sizeof(double) * (NEIGHBOR+1) *P_NUM));

	double * omega1 = new double [ome_num1];
	double * omega2 = new double [ome_num2];
	double * bias1 = new double [NEU_NUM1];
	double * bias2 = new double [NEU_NUM2];

	//随机生成Omega1
	for(int i=0; i<ome_num1; i++){
		omega1[i] = 2 * (rand()/(double)(RAND_MAX)) - 1 ;
	        if(omega1[i] == 0 || omega1[i] == -1 || omega1[i] == 1)
			omega1[i] = 0.001;
	}
	//随机生成bias1
	for(int i=0; i<NEU_NUM1; i++){
		bias1[i] = 2*(rand()/(double)(RAND_MAX)) - 1;
	}

	//随机生成Omega2
	for(int i=0; i<ome_num2; i++){
		omega2[i] = 2 * (rand()/(double)(RAND_MAX)) - 1;
		if(omega2[i] ==0 || omega2[i] == 1 || omega2[i] ==-1)
			omega2[i] = 0.001;
	}
	fprintf(stdout, "Bias1: %lf %lf %lf\n",bias1[0],bias1[1],bias1[2]);
	//随机生成bias2
	for(int i=0; i<NEU_NUM2; i++){
		bias2[i] = 2*(rand()/(double)(RAND_MAX)) - 1;
	}
	fprintf(stdout, "Bias2: %lf %lf %lf\n",bias2[0],bias2[1],bias2[2]);

	SAFE_CALL(hipMalloc((void **) &gpu_mre, sizeof(double) * mre_num * KER_NUM));//maxpooling结果存入gpu_mre，分配显存
	SAFE_CALL(hipMalloc((void **) &gpu_mre_index, sizeof(int) * mre_num * KER_NUM));//为maxpooling的最大值索引分配显存
	SAFE_CALL(hipMalloc((void **) &gpu_omega1, sizeof(double) * ome_num1));//第一层网络的输入权重，分配显存
	SAFE_CALL(hipMalloc((void **) &gpu_omega2, sizeof(double) * ome_num2));//输出层的权重，分配显存
	SAFE_CALL(hipMalloc((void **) &gpu_F1, sizeof(double) * NEU_NUM1));//第一层网络的输出，分配显存
	SAFE_CALL(hipMalloc((void **) &gpu_O2, sizeof(double) * NEU_NUM2));//输出层的结果
	SAFE_CALL(hipMalloc((void **) &gpu_bias1, sizeof(double) * NEU_NUM1));//偏置值
	SAFE_CALL(hipMalloc((void **) &gpu_bias2, sizeof(double) * NEU_NUM2));
	SAFE_CALL(hipMemcpy(gpu_omega1, omega1, sizeof(double) * ome_num1, hipMemcpyHostToDevice));//复制初始权重到GPU端
	SAFE_CALL(hipMemcpy(gpu_omega2, omega2, sizeof(double) * ome_num2, hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(gpu_bias1, bias1, sizeof(double) * NEU_NUM1, hipMemcpyHostToDevice));//复制偏置值到显存
	SAFE_CALL(hipMemcpy(gpu_bias2, bias2, sizeof(double) * NEU_NUM2, hipMemcpyHostToDevice));

	double * mre = new double [mre_num * KER_NUM];//CPU端存放maxpooling结果
	//double * F1 = new double [NEU_NUM1];//CPU端存放第一层网络输出结果
	double * O2 = new double [NEU_NUM2];//CPU端存放输出层的结果
	double * lz = new double [NEU_NUM2];
	for(int i0=0;i0<1;i0++){
		//if (i0 % 100 == 0)
		//	fprintf(stdout,"The %dst iteration.\n",i0);
		for(int j=0; j<10000000; j++){
			int iter = 0;

			//卷积，每个线程负责一个卷积核和训练数据的卷积
			convol<<<1,KER_NUM,(NEIGHBOR+1)*z*sizeof(double)>>>(iter,i0,gpu_processed_train,gpu_kernel,gpu_re,3,3,z,re_size);
			hipDeviceSynchronize();	
			//下采样，maxpooling方法，每个线程负责re的一列
			maxpooling<<<1,KER_NUM>>>(iter,gpu_re,gpu_mre,gpu_mre_index,re_size,mre_num);
			hipDeviceSynchronize();

			//全连接层
			fullconnect<<<1,NEU_NUM1,mre_size * sizeof(double)>>>(iter,gpu_mre,gpu_omega1,gpu_bias1,gpu_F1,mre_size);
			hipDeviceSynchronize();

			//输出层
			output<<<1,NEU_NUM2>>>(iter,gpu_F1,gpu_omega2,gpu_bias2,gpu_O2);
			hipDeviceSynchronize();
			SAFE_CALL(hipMemcpy(O2, gpu_O2, sizeof(double) * NEU_NUM2, hipMemcpyDeviceToHost));
			hipDeviceSynchronize();
			double loss = lossfunction(O2, processed_labels, i0);
			if(i0%100==0 && j%50==0)    fprintf(stdout,"loss:%lf \n",loss);
			if(loss < 0.001)
				break;

			//反向传播，输出层
			bp_output<<<1,NEU_NUM2>>>(iter,i0,gpu_processed_labels,gpu_O2,gpu_bias2,gpu_delta_La,gpu_delta_Lz);
			//反向传播，全连接层
			bp_fullconnect<<<1,NEU_NUM1>>>(iter,gpu_omega2,gpu_bias1,gpu_F1,gpu_delta_La,gpu_delta_Lz,gpu_delta_fa,gpu_delta_fz);
			//反向传播，maxpooling层
			bp_maxpooling<<<1,mre_size>>>(iter,mre_size,gpu_mre_index,gpu_omega1,gpu_mre,gpu_delta_fa,gpu_delta_fz,gpu_delta_ma,gpu_delta_2,gpu_delta_22);
			//反向传播，map到卷积层
			//bp_map_convol<<<1,mre_size>>>(iter,mre_num,gpu_mre_index,gpu_re,gpu_delta_2,gpu_delta_22);
			bp_update_kernel<<<1,KER_NUM,(NEIGHBOR+1)*z*sizeof(double)>>>(iter,i0,3,3,z,mre_num,re_size,gpu_mre_index,gpu_delta_22,gpu_processed_train,gpu_delta_kw,gpu_kernel);

			hipDeviceSynchronize();
			SAFE_CALL(hipMemcpy(bias2, gpu_bias2, sizeof(double) * NEU_NUM2, hipMemcpyDeviceToHost));
			/*if(j <5){
				fprintf(stdout,"bias2:\n ");
				for(int j=0; j<16; j++)
					fprintf(stdout," %lf\n",bias2[j]);
				fprintf(stdout,"\n");
			}*/
		}
	}
	int * mre_index = new int [mre_size];

	fprintf(stdout,"Training completed!\n");
	//hipDeviceSynchronize();
	//SAFE_CALL(hipMemcpy(re, gpu_re, sizeof(double) * re_size * KER_NUM, hipMemcpyDeviceToHost));
	//SAFE_CALL(hipMemcpy(mre,gpu_mre,sizeof(double) * mre_num * KER_NUM, hipMemcpyDeviceToHost));
	//SAFE_CALL(hipMemcpy(mre_index,gpu_mre_index,sizeof(int) * mre_size, hipMemcpyDeviceToHost));
	SAFE_CALL(hipMemcpy(O2,gpu_O2,sizeof(double) * NEU_NUM2, hipMemcpyDeviceToHost));
	//SAFE_CALL(hipMemcpy(omega1, gpu_omega1, sizeof(double) * ome_num1, hipMemcpyDeviceToHost));
	//SAFE_CALL(hipMemcpy(omega2, gpu_omega2, sizeof(double) * ome_num2, hipMemcpyDeviceToHost));
	hipDeviceSynchronize();

	//fprintf(stdout,"result:%lf %lf %lf\n",re[0],re[5],re[10]);
	//fprintf(stdout,"result:%lf %lf %lf\n",re[1],re[6],re[11]);
	//fprintf(stdout,"result:%lf %lf %lf\n",re[2],re[7],re[12]);
	//fprintf(stdout,"result:%lf %lf %lf\n",re[3],re[8],re[13]);
	//fprintf(stdout,"result:%lf %lf %lf\n",re[4],re[9],re[14]);

	//fprintf(stdout,"mre:%lf %lf %lf\n",mre[0],mre[1],mre[2]);
	//fprintf(stdout,"mre_index:%d %d %d\n",mre_index[0],mre_index[1],mre_index[2]);

	//fprintf(stdout,"F1 Output:%lf %lf; %lf %lf\n",F1[0],F1[1],F1[98],F1[99]);
	fprintf(stdout,"O2 Output:%lf %lf; %lf %lf\n",O2[0],O2[1],O2[18],O2[19]);
	
	return 0;
}

//主函数
int main(int argc, char * argv[])
{
  	if(!InitCUDA()){
		return 0;
	}
	printf("CUDA initialized.\n");

	clock_t start,end;

	double *trainset,*trainlabels;
	if(argc!=2){
		fprintf(stderr, "4 input arguments required!");
	}
	MATFile * datamat = matOpen(argv[1], "r");
	mxArray * train = matGetVariable(datamat,"DataSet");
	mxArray * labels = matGetVariable(datamat,"labels");

	trainset = (double*)mxGetData(train);
	trainlabels = (double*)mxGetData(labels);

	const mwSize  * dim;
	dim = mxGetDimensions(train);//获取trainset每维的元素个数

	start = clock();
	int te = training(trainset, trainlabels, dim[0], dim[1], dim[2]);
	end = clock();
	double usetime = double(end - start);
	fprintf(stdout, "Using time of preprocessing:%lfs\n",usetime/CLOCKS_PER_SEC);
	return 0;
}
